#include "hip/hip_runtime.h"
#include <algorithm> // std::fill_n
#include <iostream>  // snprintf
#include <math.h>    // expf, log
#include <stdlib.h>  // rand
#include <string>    // std::string
#include <vector>    // std::vector

#include "src/kernels/fused_decoder_self_attention.h"
#include "src/utils/macro.h"

// bug1: MUST add CHECK to hipMemcpy to see if its work well
template <typename T>
void CPUMaskedAttn(T *q,
                   T *k,
                   T *v,
                   T *k_cache,
                   T *v_cache,
                   T *mha_output,
                   const int batch_size,
                   const int num_heads,
                   const int head_size,
                   const int step)
{
    int batch_stride = num_heads * head_size;
    int head_stride = head_size;
    int cache_offset = batch_size * batch_stride;
    int block_nums = batch_size * num_heads;
    float scale = rsqrt(float(head_size));

    const T *q_mem = q;
    const T *k_mem = k;
    const T *v_mem = v;

    // tmp buffer
    float *sqk = (float *)malloc(sizeof(float) * (block_nums * (3 * head_size + step)));
    float *sq = sqk;
    float *sk = sq + block_nums * head_size;
    float *logits = sk + block_nums * head_size;
    float *sv = logits + block_nums * step;
    // FT 2.1的写法里面，kv cache是在prompt阶段已经填充，iter=0为token gen的起始iter
    for (int batch_id = 0; batch_id < batch_size; batch_id++)
    {
        for (int head_id = 0; head_id < num_heads; head_id++)
        {
            float row_max = 0.0f;
            for (int iter = 0; iter < step; iter++)
            {
                float attn_score = 0.0f;
                for (int tid = 0; tid < head_size; tid++)
                {
                    int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
                    // note: sq and sk's offset should be qkv_offset , not tid
                    sk[qkv_offset] = k_cache[iter * cache_offset + qkv_offset];
                    // when final step, update k cache
                    if (iter == step - 1)
                    {
                        // TODO: update k cache with k with bias add
                        k_cache[iter * cache_offset + qkv_offset] = (float)k_mem[qkv_offset];
                        sk[qkv_offset] = (float)k_mem[qkv_offset];
                    }

                    sq[qkv_offset] = (float)q_mem[qkv_offset];
                    float qk = sq[qkv_offset] * sk[qkv_offset] * scale;
                    // block reduce using multi warp reduce
                    // TODO: maybe broadcast the attn score to each thread of the block in blockreducesum
                    attn_score += qk;
                }
                // note: logtis's offset should be as follow, not should mul head size with iter
                // debug info,printf("every step/seqlen attn score = %f\n", attn_score);
                logits[batch_id * num_heads * step + head_id * step + iter] = attn_score;
                // softmax(logits), logits.shape = [bs, num heads, 1, step]
                row_max = std::max(attn_score, row_max);
            }
            printf("all step/seqlen(one row) max attn score = %f\n", row_max);
            float fenzi = 0.0f;
            float fenmu = 0.0f;
            for (int iter = 0; iter < step; iter++)
            { // row
                fenzi = expf(logits[batch_id * num_heads * step + head_id * step + iter] - row_max);
                fenmu += fenzi;
            }
            for (int iter = 0; iter < step; iter++)
            { // row
                logits[batch_id * num_heads * step + head_id * step + iter] = fenzi / fenmu;
                printf("logits=%f\n", fenzi / fenmu);
            }
            // logits*V = [bs, num heads, 1, step] * [mx_seq_len or step, bs, num heads, head size]
            // for(int iter = 0; iter < step; iter++) {
            for (int tid = 0; tid < head_size; tid++)
            {
                float O = 0.0f;
                int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
                for (int iter = 0; iter < step; iter++)
                {
                    sv[qkv_offset] = v_cache[iter * cache_offset + qkv_offset];
                    // when final step, update k cache
                    if (iter == step - 1)
                    {
                        // TODO: update k cache with k with bias add
                        v_cache[iter * cache_offset + qkv_offset] = (float)v_mem[qkv_offset];
                        sv[qkv_offset] = (float)v_mem[qkv_offset];
                    }
                    O += sv[qkv_offset] * logits[batch_id * num_heads * step + head_id * step + iter];
                    printf("logits[%d]=%f, sv[%d]=%f, O=%f\n", iter, logits[iter], qkv_offset, sv[qkv_offset], O);
                }
                mha_output[qkv_offset] = O;
            }
        }
    }

    free(sqk);
}
template <typename T>
bool CheckResult(float *CPUoutput, T *GPUoutput, int output_size)
{
    for (int i = 0; i < output_size; i++)
    {
        float GPUres = (float)GPUoutput[i];
        if (fabs(CPUoutput[i] - GPUres) > 1e-6)
        {
            printf("the %dth res is wrong, CPUoutput = %f, GPUoutput = %f\n", i, CPUoutput[i], GPUres);
            return false;
        }
    }
    return true;
}

#define LAUNCH_FUSED_ATTN(dtype)                                                                                                      \
    dtype *h_qkv;                                                                                                                     \
    dtype *d_qkv;                                                                                                                     \
    int qkv_size = batch_size * (2 * kv_num_heads + num_heads) * head_size;                                                           \
    h_qkv = (dtype *)malloc(sizeof(dtype) * qkv_size);                                                                                \
    hipMalloc((void **)&d_qkv, sizeof(dtype) * qkv_size);                                                                            \
    dtype *h_kcache;                                                                                                                  \
    dtype *d_kcache;                                                                                                                  \
    int kcache_size = max_seq_len * batch_size * kv_num_heads * head_size;                                                            \
    h_kcache = (dtype *)malloc(sizeof(dtype) * kcache_size);                                                                          \
    hipMalloc((void **)&d_kcache, sizeof(dtype) * kcache_size);                                                                      \
    dtype *h_vcache;                                                                                                                  \
    dtype *d_vcache;                                                                                                                  \
    int vcache_size = max_seq_len * batch_size * kv_num_heads * head_size;                                                            \
    h_vcache = (dtype *)malloc(sizeof(dtype) * vcache_size);                                                                          \
    hipMalloc((void **)&d_vcache, sizeof(dtype) * vcache_size);                                                                      \
    for (int i = 0; i < qkv_size; i++)                                                                                                \
    {                                                                                                                                 \
        h_qkv[i] = (dtype)1.0f;                                                                                                       \
    }                                                                                                                                 \
    dtype *h_q = h_qkv;                                                                                                               \
    dtype *h_k = h_q + batch_size * num_heads * head_size;                                                                            \
    dtype *h_v = h_k + batch_size * (kv_num_heads + num_heads) * head_size;                                                           \
    for (int i = 0; i < (kcache_size * h_step) / max_seq_len; i++)                                                                    \
    {                                                                                                                                 \
        h_kcache[i] = (dtype)1.0f;                                                                                                    \
        h_vcache[i] = (dtype)1.0f;                                                                                                    \
    }                                                                                                                                 \
    dtype *h_o;                                                                                                                       \
    dtype *d_o;                                                                                                                       \
    int o_size = batch_size * num_heads * head_size;                                                                                  \
    h_o = (dtype *)malloc(sizeof(dtype) * o_size);                                                                                    \
    hipMalloc((void **)&d_o, sizeof(dtype) * o_size);                                                                                \
    bool *h_finished = (bool *)malloc(sizeof(bool) * batch_size);                                                                     \
    bool *d_finished;                                                                                                                 \
    for (int i = 0; i < batch_size; i++)                                                                                              \
    {                                                                                                                                 \
        h_finished[i] = static_cast<bool>(0);                                                                                         \
    }                                                                                                                                 \
    dtype *h_qkv_bias = (dtype *)malloc(sizeof(dtype) * (2 * kv_num_heads + num_heads) * head_size);                                  \
    dtype *d_qkv_bias;                                                                                                                \
    hipMalloc((void **)&d_qkv_bias, sizeof(dtype) * (2 * kv_num_heads + num_heads) * head_size);                                     \
    for (int i = 0; i < (2 * kv_num_heads + num_heads) * head_size; i++)                                                              \
    {                                                                                                                                 \
        h_qkv_bias[i] = (dtype)0.0f;                                                                                                  \
    }                                                                                                                                 \
    hipMemcpy(d_qkv, h_qkv, sizeof(dtype) * batch_size * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);        \
    hipMemcpy(d_qkv_bias, h_qkv_bias, sizeof(dtype) * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);           \
    hipMemcpy(d_finished, h_finished, sizeof(bool) * batch_size, hipMemcpyHostToDevice);                                            \
    hipMemcpy(d_kcache, h_kcache, sizeof(dtype) * kcache_size, hipMemcpyHostToDevice);                                              \
    hipMemcpy(d_vcache, h_vcache, sizeof(dtype) * vcache_size, hipMemcpyHostToDevice);                                              \
    DataType type = getTensorType<dtype>();                                                                                           \
    DataType type_bool = getTensorType<bool>();                                                                                       \
    DataType type_int = getTensorType<int>();                                                                                         \
    TensorWrapper<dtype> *qkv = new TensorWrapper<dtype>(GPU, type, {batch_size, num_heads + 2 * kv_num_heads, head_size}, d_qkv);    \
    TensorWrapper<dtype> *kcache = new TensorWrapper<dtype>(GPU, type, {max_seq_len, batch_size, kv_num_heads, head_size}, d_kcache); \
    TensorWrapper<dtype> *vcache = new TensorWrapper<dtype>(GPU, type, {max_seq_len, batch_size, kv_num_heads, head_size}, d_vcache); \
    TensorWrapper<bool> *finished = new TensorWrapper<bool>(GPU, type_bool, {batch_size}, d_finished);                                \
    TensorWrapper<int> *step = new TensorWrapper<int>(CPU, type_int, {1}, &h_step);                                                   \
    TensorWrapper<int> *layer_id = new TensorWrapper<int>(CPU, type_int, {1}, &h_layer_id);                                           \
    TensorWrapper<dtype> *mha_output = new TensorWrapper<dtype>(GPU, type, {batch_size, num_heads, head_size}, d_o);                  \
    BaseWeight<dtype> qkv_weight;                                                                                                     \
    qkv_weight.bias = d_qkv_bias;                                                                                                     \
    LLaMAAttentionStaticParams params;                                                                                                \
    params.rotary_embedding_dim = rotary_embedding_dim;                                                                               \
    params.rotary_embedding_base = rotary_embedding_base;                                                                             \
    params.max_position_embeddings = max_position_embeddings;                                                                         \
    params.use_dynamic_ntk = false;                                                                                                   \
    launchDecoderMaskedMHA(qkv, qkv_weight, layer_id, kcache, vcache, finished, step, mha_output, params);                     \
    CHECK(hipMemcpy(h_o, d_o, sizeof(dtype) * o_size, hipMemcpyDeviceToHost));                                                      \
    float *CPU_output = (float *)malloc(sizeof(float) * o_size);                                                                      \
    CPUMaskedAttn<dtype>(h_q, h_k, h_v, h_kcache, h_vcache, CPU_output, batch_size, num_heads, head_size, h_step);                      \
    bool is_true = CheckResult<dtype>(CPU_output, h_o, o_size);                                                                       \
    if (is_true)                                                                                                                      \
    {                                                                                                                                 \
        printf("test passed");                                                                                                        \
    }                                                                                                                                 \
    else                                                                                                                              \
    {                                                                                                                                 \
        printf("test failed");                                                                                                        \
    }                                                                                                                                 \
    free(h_qkv);                                                                                                                      \
    free(h_kcache);                                                                                                                   \
    free(h_vcache);                                                                                                                   \
    free(h_o);                                                                                                                        \
    free(CPU_output);                                                                                                                 \
    free(h_finished);                                                                                                                 \
    hipFree(d_finished);                                                                                                             \
    hipFree(d_qkv);                                                                                                                  \
    hipFree(d_o);                                                                                                                    \
    hipFree(d_kcache);                                                                                                               \
    hipFree(d_vcache);

int main(int argc, char *argv[])
{
    constexpr int batch_size = 1;
    constexpr int head_size = 16;
    constexpr int num_heads = 2;
    constexpr int kv_num_heads = 1;
    constexpr int max_seq_len = 32;
    int h_step = 4;
    int h_layer_id = 0;
    int rotary_embedding_dim = 128;
    float rotary_embedding_base = 10000;
    int max_position_embeddings = 2048;
    bool use_dynamic_ntk = false; // for dyn scaling rope
    if (argv[1])
    {
        LAUNCH_FUSED_ATTN(half);
    }
    else
    {
        LAUNCH_FUSED_ATTN(float);
    }
}
