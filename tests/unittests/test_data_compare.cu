#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector
#include <stdio.h>
#include <fstream>
#include "src/utils/macro.h"
#include "src/utils/debug_utils.h"
// (RussWong)note:
// this test is for debug, to compare intermediate tensor and HF intermediate tensor
// before run, you should change the path to your local right dir
// `./debug` to compare 

std::vector<float> loadWeightFromBinHelper(std::vector<size_t> shape, std::string filename)
{
    size_t dim0 = 1, dim1 = 1;
    if (shape.size() > 2) {
        dim0 = shape[0] * shape[1];
        dim1 = shape[2];
    }

    if (shape.size() == 2) {
        dim0 = shape[0];
        dim1 = shape[1];
    }
    size_t size = dim0 * dim1;
    if (size == 0) {
        std::cout << "shape is zero, skip loading weight from file: " << filename << std::endl;
        return std::vector<float>();
    }

    std::vector<float> host_array(size);
    std::ifstream  in(filename, std::ios::in | std::ios::binary);
    if (!in.is_open()) {
        std::cout << "file" << filename << "cannot be opened, loading model fails!" << std::endl;
        return std::vector<float>();
    }

    size_t loaded_data_size = sizeof(float) * size;
    in.seekg(0, in.end);
    in.seekg(0, in.beg);

    std::cout << "Read " << std::to_string(loaded_data_size) << " bytes from " << filename << std::endl;
    in.read((char*)host_array.data(), loaded_data_size);

    size_t in_get_size = in.gcount();
    if (in_get_size != loaded_data_size) {
        return std::vector<float>();
    }
    in.close();
    // If we succeed, return an array with values.
    return host_array;
}
void internalFunc(float* ptr, std::vector<size_t> shape, std::string filename) {
    std::vector<float> host_array = loadWeightFromBinHelper(shape, filename);
    if (host_array.empty()) {
        std::cout << "[warning] data from file is empty!!" << "\n";
        return;
    }
    // copy host_array to our defined ptr
    memcpy(ptr, host_array.data(), host_array.size());
    return;
}
void loadWeights(float* ptr1, std::string weight_path, int shape0, int shape1) // weighttype参数比较多余
{
    // load attn output
    internalFunc(ptr1, {(size_t)shape0, (size_t)shape1}, weight_path);

}
void loadWeights_trans(float* ptr1, std::string weight_path, int shape0, int shape1) // weighttype参数比较多余
{
    // load attn output
    internalFunc(ptr1, {(size_t)shape0, (size_t)shape1}, weight_path);

}

bool CheckResult(float* CPUoutput, float* GPUoutput, int output_size) {
    for(int i = 0; i < output_size; i++) {
        
	if(fabs(CPUoutput[i] - GPUoutput[i]) > 1e-6){
	    printf("the %dth res is wrong, onellm = %f, trans = %f\n", i, CPUoutput[i], GPUoutput[i]);
    }
    return true;
}
// 1.for example: the path of two data files is below, and you should replace L122&L123 with the two
// /home/data/trans/q_buf_after_rope_trans.bin
// /home/data/onellm/q_buf_after_rope.bin
// 2.And you should change the L98&L99 to the right data size according to your data file
int main(int argc, char *argv[]) {
    const int seqlen = 13;
    const int hidden_units = 4096;
    const int vocab_size = 32;
    const int inter_size = 10;
    int hidden_units_2 = 0;
    int output_size = 0;
    int shape0 = 1; // TO MODIFY
    int shape1 = 4096; // TO MODIFY
    
    int in_size = shape0 * shape1;
    hidden_units_2 = hidden_units * hidden_units;
    output_size = seqlen * hidden_units;
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    float* h_w;
    float* d_w = (float*)malloc(sizeof(float) * hidden_units_2);
    float* d_w_trans= (float*)malloc(sizeof(float) * hidden_units_2);
    h_w = (float*)malloc(sizeof(float) * hidden_units_2);
    for(int i = 0; i < hidden_units_2; i++) { 
       h_w[i] = (float)(i % 3); // 1 2 1 2
    }

    float* h_in = (float*) malloc(sizeof(float) * hidden_units * seqlen);
    float* d_in = (float*) malloc(sizeof(float) * in_size);
    float* d_in_trans = (float*) malloc(sizeof(float) * in_size);
    for(int i = 0; i < hidden_units * seqlen; i++) { 
       h_in[i] = (float)(i % 3);
    }

    float* h_out = (float*) malloc(sizeof(float) * output_size);
    float* d_out;
    hipMalloc((void**)&d_out, sizeof(float) * output_size);
    loadWeights(d_in, "/home/data/onellm/0_self_decoder_qk_v_after_bmm.bin", shape0, shape1); // TO MODIFY
    loadWeights_trans(d_in_trans, "/home/data/trans/self_decoder_qk_v_buf_after_bmm_trans.bin", shape0, shape1); // TO MODIFY
    std::cout << "====intermediate tensor====" << "\n";
    CheckResult(d_in, d_in_trans, shape0 * shape1);

    free(h_in);
    free(h_w);
    free(h_out);
    free(d_in);
    free(d_w);
    free(d_in_trans);
    free(d_w_trans);
    hipFree(d_out);
}
