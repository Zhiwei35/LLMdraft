#include "hip/hip_runtime.h"
#include <float.h> //FLT_MIN
#include <hip/hip_runtime.h>
#include <iostream>
//#include <utils/gpu_config.h>
#include "src/kernels/topK.h"
#include <hipcub/hipcub.hpp>

// Note: a b两个topK reduce输出一个topK
template<int K>
__device__ topK<K> reduce_functor(const topK<K>& a, const topK<K>& b) {
    topK<K> res = a;
    for(int i = 0; i < K; i++){
        res.insertHeap(b.val[i], b.id[i]);
    }
    return res;
}
// gridsize:bs * beamwidth * BlockPerBeam 
// blocksize:256
// shape infer: [bs, beamwidth, vocab size] => [bs, beamwidth, BlockPerBeam, K]
template<int K, int blockSize, int BlockPerBeam>
__global__ void topK_kernel_round1(const float* probs, const int vocab_size, 
                                         int* topK_ids, float* topK_vals)
{
    typedef hipcub::BlockReduce<topK<K>, blockSize> blockreduce;
    __shared__ typename blockreduce::TempStorage temp_storage;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int row_id = bid / BlockPerBeam;
    int block_lane = bid % BlockPerBeam;
    topK<K> thread_topK;
    thread_topK.init();
    // thread local reduce
    for(int data_id = tid + block_lane * blockSize; data_id < vocab_size; data_id += BlockPerBeam * blockSize){
        int data_offset = data_id + row_id * vocab_size;
        float data = probs[data_offset];
        thread_topK.insertHeap(data, data_offset);
       //thread_topK.insertHeap(data, data_id); // bug
    }
    //block local reduce
    topK<K> block_topK = blockreduce(temp_storage).Reduce(thread_topK, reduce_functor<K>);

    if(tid == 0){
        for(int k_offset = 0; k_offset < K; k_offset++) {
            // topK_vals[row_id * vocab_size + block_lane * blockSize + k_offset] = block_topK.val[k_offset]; //bug
            topK_vals[row_id * BlockPerBeam * K + block_lane * K + k_offset] = block_topK.val[k_offset];
            topK_ids[row_id * BlockPerBeam * K  + block_lane * K + k_offset] = block_topK.id[k_offset];//output offset要根据output buffer的shape来计算

        }
    }
}
// shape infer: [bs, beamwidth, BlockPerBeam, K] => [bs, K]
// ids是beamwidth * vocalsize中的全局word id
// gridSize = bs
// blockSize = 256
template<int K, int blockSize, int BlockPerBeam>
__global__ void topK_kernel_round2(const int* topK_ids, const float* topK_vals,
                                    int* final_topK_ids, float* final_topK_vals)
{
    typedef hipcub::BlockReduce<topK<K>, blockSize> blockreduce;
    __shared__ typename blockreduce::TempStorage temp_storage;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int row_id = bid;
    topK<K> thread_topK;
    // thread local reduce    
    for(int i = tid; i < BlockPerBeam * K; i += blockDim.x) {
        int data_offset = bid * BlockPerBeam * K + i;
        thread_topK.insertHeap(topK_vals[data_offset], topK_ids[data_offset]);
    }
    // block reduce
    topK<K> block_topK = blockreduce(temp_storage).Reduce(thread_topK, reduce_functor<K>);
    if(tid == 0){
        for(int k_offset = 0; k_offset < K; k_offset++) {
            // topK_vals[row_id * vocab_size + block_lane * blockSize + k_offset] = block_topK.val[k_offset]; //bug
            final_topK_vals[bid * K + k_offset] = block_topK.val[k_offset];
            final_topK_ids[bid * K + k_offset] = block_topK.id[k_offset];
        }
    }    
}

void launchTopKforBeamSearch(const float* probs, 
                            const int batch_size,
                            const int vocab_size, 
                            int* topk_ids,
			    float* topk_vals,
			    int* final_topk_ids,
			    float* final_topk_vals) {// GPU workspace is for intermediate buffer and output buffer
    constexpr int BlockPerBeam = 8;
    constexpr int beamwidth = 1;
    constexpr int K = 5;
    // buffer size
    int topK_val_buf_size = batch_size * beamwidth * BlockPerBeam * K;
    int topK_ids_buf_size = batch_size * beamwidth * BlockPerBeam * K;
    int final_topK_val_buf_size = batch_size * beamwidth * K;
    // memory plan
    float* topK_vals = topk_vals;
    int* topK_ids = topk_ids;
    float* final_topK_vals = final_topk_vals;
    int* final_topK_ids = final_topk_ids;    
    // prepare launch
    // TODO: add GPUconfig API to easily get GPU config, ep: maxblocknums
    // GPUConfig config;
    // int maxBlockNums = config.getMaxBlockNums();
    // TODO: how to alloc block nums more flexable according to shape
    //constexpr int BlockPerBeam = 8;
    int maxBlockNums = 1024;
    int BlockNums1 = std::min(batch_size * beamwidth * BlockPerBeam, maxBlockNums);
    int BlockNums2 = std::min(batch_size * beamwidth, maxBlockNums);
    dim3 grid_round1(BlockNums1);
    dim3 block_round1(256);
    dim3 grid_round2(BlockNums2);
    dim3 block_round2(256);
    // debug info, better to retain: std::cout << "in cu file, before launch" << std::endl;
    topK_kernel_round1<K, 256, BlockPerBeam>
                        <<<grid_round1, block_round1>>>(probs, vocab_size, topK_ids, topK_vals);
    topK_kernel_round2<K, 256, BlockPerBeam>
                        <<<grid_round2, block_round2>>>(topK_ids, topK_vals, final_topK_ids, final_topK_vals);
    // debug info, better to retain: std::cout << "in cu file, after launch" << std::endl;
}
