#include "hip/hip_runtime.h"
#include <iostream>
#include "src/kernels/act_kernel.h"

template<typename T>
__device__ __forceinline__ T silu(const T& in) {
  // x * sigmoid(x)
  return (T) (((float) in) / (1.0f + expf((float) -in)));
}

//第一个intermediate size去做silu，结果与第二个intermediate mul
template<typename T>
__global__ void silu_and_mul_kernel(
  T* out,               // [bs, intermedia size]
  const T* input,       // [bs, 2, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  for (int idx = threadIdx.x; idx < intermedia_size; idx += blockDim.x) {
    const T x = input[batch_idx * 2 * intermedia_size + idx];
    const T y = input[batch_idx * 2 * intermedia_size + intermedia_size + idx];
    out[batch_idx * intermedia_size + idx] = silu<T>(x) * y;
  }
}

template<typename T>
void launchAct(TensorWrapper<T>* input, TensorWrapper<T>* out) {
    int batch_size = input->shape[1];
    int intermedia_size = input->shape[2];
    dim3 grid(batch_size);
    dim3 block(256);
    // std::cout << "calling silu_and_mul kernel" << "\n";
    silu_and_mul_kernel<T><<<grid, block>>>(out->data, input->data, intermedia_size);
    // std::cout << "called silu_and_mul kernel" << "\n";
}
// We must instancite the template, if not, will report linking issue
template void launchAct(TensorWrapper<float>* input, TensorWrapper<float>* output);
template void launchAct(TensorWrapper<half>* input, TensorWrapper<half>* output);
