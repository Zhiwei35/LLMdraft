#include "hip/hip_runtime.h"
#include <iostream>
#include "src/kernels/act_kernel.h"
#include "src/utils/cuda_debug_utils.cuh"
#include "src/utils/macro.h"
template<typename T>
__device__ __forceinline__ T silu(const T& in) {
  // x * sigmoid(x)
  return (T) (((float) in) / (1.0f + expf((float) -in)));
}

template<>
__device__ __forceinline__ half2 silu<half2>(const half2& in) {
  return make_half2(__float2half(silu<float>((float)(in.x))), __float2half(silu<float>((float)(in.y))));
}

//代码逻辑：第一个intermediate 去做silu，结果与第二个intermediate mul
template<typename T>
__global__ void silu_and_mul_kernel(
  T* out,               // [bs, intermedia size]
  const T* input,       // [bs, 2, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  for (int idx = threadIdx.x; idx < intermedia_size; idx += blockDim.x) { 
    const T x = input[batch_idx * 2 * intermedia_size + idx];
    const T y = input[batch_idx * 2 * intermedia_size + intermedia_size + idx];
    out[batch_idx * intermedia_size + idx] = silu<T>(x) * y;
  }
}

template<>
__global__ void silu_and_mul_kernel<half>(
  half* out,               // [bs, intermedia size]
  const half* input,       // [bs, 2, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  int vec_size = Vec<half>::size;
  using Vec_t = typename Vec<half>::Type;
  for (int idx = threadIdx.x * vec_size; idx < intermedia_size; idx += blockDim.x) {
    const Vec_t x = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[batch_idx * 2 * intermedia_size + idx]));
    const Vec_t y = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[batch_idx * 2 * intermedia_size + intermedia_size + idx]));
    *reinterpret_cast<Vec_t*>(&out[batch_idx * intermedia_size + idx]) = __hmul2(silu<Vec_t>(x), y);
  }

}

template<typename T>
void launchAct(TensorWrapper<T>* input, TensorWrapper<T>* out) {
    int batch_size = input->shape[0];
    LLM_CHECK(input->shape[1] == 2);
    int intermedia_size = input->shape[2];
    dim3 grid(batch_size);
    dim3 block(256);
    silu_and_mul_kernel<T><<<grid, block>>>(out->data, input->data, intermedia_size);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(out->data);
#else
#endif
}
// We must instancite the template, if not, will report linking issue
template void launchAct(TensorWrapper<float>* input, TensorWrapper<float>* output);
template void launchAct(TensorWrapper<half>* input, TensorWrapper<half>* output);
