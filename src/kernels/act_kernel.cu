#include "hip/hip_runtime.h"
#include <iostream>
#include "src/kernels/act_kernel.h"

template<typename T>
__device__ __forceinline__ T silu(const T& in) {
  // x * sigmoid(x)
  return (T) (((float) in) / (1.0f + expf((float) -in)));
}

template<>
__device__ __forceinline__ half2 silu<half2>(const half2& in) {
  return make_half2(__float2half(silu<float>((float)(in.x))), __float2half(silu<float>((float)(in.y))));
}
//第一个intermediate size去做silu，结果与第二个intermediate mul
template<typename T>
__global__ void silu_and_mul_kernel(
  T* out,               // [bs, intermedia size]
  const T* input,       // [bs, 2, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  for (int idx = threadIdx.x; idx < intermedia_size; idx += blockDim.x) {
    const T x = input[batch_idx * 2 * intermedia_size + idx];
    const T y = input[batch_idx * 2 * intermedia_size + intermedia_size + idx];
    out[batch_idx * intermedia_size + idx] = silu<T>(x) * y;
  }
}

template<>
__global__ void silu_and_mul_kernel<half>(
  half* out,               // [bs, intermedia size]
  const half* input,       // [2, bs, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  int vec_size = Vec<half>::size;
  using Vec_t = typename Vec<half>::Type;
  // Vec_t x_vec; 
  for (int idx = threadIdx.x * vec_size; idx < intermedia_size; idx += blockDim.x) {
    const Vec_t x = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[batch_idx * 2 * intermedia_size + idx]));
    const Vec_t y = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[batch_idx * 2 * intermedia_size + intermedia_size + idx]));
    *reinterpret_cast<Vec_t*>(&out[batch_idx * intermedia_size + idx]) = __hmul2(silu<Vec_t>(x), y);
  }
}

template<typename T>
void launchAct(TensorWrapper<T>* input, TensorWrapper<T>* out) {
    int batch_size = input->shape[1];
    int intermedia_size = input->shape[2];
    dim3 grid(batch_size);
    dim3 block(256);
    // std::cout << "calling silu_and_mul kernel" << "\n";
    silu_and_mul_kernel<T><<<grid, block>>>(out->data, input->data, intermedia_size);
    // std::cout << "called silu_and_mul kernel" << "\n";
}
// We must instancite the template, if not, will report linking issue
template void launchAct(TensorWrapper<float>* input, TensorWrapper<float>* output);
template void launchAct(TensorWrapper<half>* input, TensorWrapper<half>* output);
